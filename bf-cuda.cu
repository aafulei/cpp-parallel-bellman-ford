#include "hip/hip_runtime.h"
/*
* This is a CUDA version of bellman_ford algorithm
* Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_bellman_ford cuda_bellman_ford.cu
* Run: ./cuda_bellman_ford <input file> <number of blocks per grid> <number of threads per block>, you will find the output file 'output.txt'
* */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <iomanip>
#include <cstring>
#include <ctime>


#include <hip/hip_runtime.h>
#include <>

#include "pnt.hpp"

using std::string;
using std::cout;
using std::endl;

#define INF 1000000


void pprint(int i, int n, bool stay = false)
{
	int p = (i + 1) * 100 / n;
	if (p != i * 100 / n)
		printf("%d%%\r", p);
	if (stay && p == 100)
		putchar('\n');
}

/*
* This is a CHECK function to check CUDA calls
*/
#define CHECK(call)                                                            \
		{                                                                              \
	const hipError_t error = call;                                            \
	if (error != hipSuccess)                                                  \
	{                                                                          \
		fprintf(stderr, "Error: %s%d, ", __FILE__, __LINE__);                 \
		fprintf(stderr, "code: %d, reason: %s\n", error,                       \
				hipGetErrorString(error));                                    \
				exit(1);                                                               \
	}                                                                          \
		}


/**
* utils is a namespace for utility functions
* including I/O (read input file and print results) and matrix dimension convert(2D->1D) function
*/
namespace utils {
	int N; //number of vertices
	int *mat; // the adjacency matrix

	void abort_with_error_message(string msg) {
		std::cerr << msg << endl;
		abort();
	}

	//translate 2-dimension coordinate to 1-dimension
	int convert_dimension_2D_1D(int x, int y, int n) {
		return x * n + y;
	}

	int read_file(string filename) {
		std::ifstream inputf(filename, std::ifstream::in);
		if (!inputf.good()) {
			abort_with_error_message("ERROR OCCURRED WHILE READING INPUT FILE");
		}
		inputf >> N;
		//input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
		assert(N < (1024 * 1024 * 20));
		mat = (int *)malloc(N * N * sizeof(int));
		printf("%d int malloced\n", N * N);
		for (int i = 0; i < N; i++) {
			for (int j = 0; j < N; j++) {
				inputf >> mat[convert_dimension_2D_1D(i, j, N)];
			}
			pprint(i, N);
		}
		return 0;
	}

	int print_result(bool has_negative_cycle, int *dist) {
		std::ofstream outputf("output.txt", std::ofstream::out);
		if (!has_negative_cycle) {
			for (int i = 0; i < N; i++) {
				if (dist[i] > INF)
					dist[i] = INF;
				outputf << dist[i] << '\n';
			}
			outputf.flush();
		}
		else {
			outputf << "FOUND NEGATIVE CYCLE!" << endl;
		}
		outputf.close();
		return 0;
	}
} //namespace utils

 // you may add some helper/kernel functions here.

__global__ void relax_initial(int * d_dist, bool * d_has_negative_cycle, bool * relaxed_last_round, bool * relaxed_this_round, int * relaxed_times, int n)
{
	int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
	int i = bdim * bid + tid;
	int skip = bdim * gdim;
	for (int k = i; k < n; k += skip) {
		d_dist[k] = INF;
		relaxed_last_round[k] = false;
		relaxed_this_round[k] = false;
		relaxed_times[k] = 0;
	}

	if (i == 0) {
		d_dist[0] = 0;
		*d_has_negative_cycle = false;// changed this morning, you forget * last night
		relaxed_last_round[0] = true;
	}
	__syncthreads();
}

__global__ void relax_swap(bool * relaxed_last_round, bool * relaxed_this_round, int n)
{
	int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
	int i = bdim * bid + tid;
	int skip = bdim * gdim;

	for (int j = i; j < n; j += skip) {
		relaxed_last_round[j] = relaxed_this_round[j];
		relaxed_this_round[j] = false;
	}
	__syncthreads();
}

__global__ void bf(int n, int const* d_mat, int * d_dist, bool * d_has_change, bool * d_has_negative_cycle, bool const* relaxed_last_round, bool * relaxed_this_round, int * relaxed_times)
{
	int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
	int i = bdim * bid + tid;
	int skip = bdim * gdim;

	if (i == 0)
		*d_has_change = false;
	__syncthreads();

	bool my_has_change = false;

	for (int v = i; v < n; v += skip) {
		for (int u = 0; u < n; ++u) {
			if (relaxed_last_round[u]) {
				int weight = d_mat[u * n + v];
				if (weight < INF)
					if (d_dist[u] + weight < d_dist[v]) {
						d_dist[v] = d_dist[u] + weight;
						relaxed_times[v] += 1;
						relaxed_this_round[v] = true;
						my_has_change = true;
						if (v == 0 && d_dist[v] < 0)
							*d_has_negative_cycle = true;
						if (relaxed_times[v] == n)
							*d_has_negative_cycle = true;
					}
			}
		}
	}
	if (my_has_change)
		*d_has_change = true;
}

/**
* Bellman-Ford algorithm. Find the shortest path from vertex 0 to other vertices.
* @param blockPerGrid number of blocks per grid
* @param threadsPerBlock number of threads per block
* @param n input size
* @param *mat input adjacency matrix
* @param *dist distance array
* @param *has_negative_cycle a bool variable to recode if there are negative cycles
*/
void bellman_ford(int blocksPerGrid, int threadsPerBlock, int n, int *mat, int *dist, bool *has_negative_cycle) {
	//------your code starts from here-----
	dim3 gdim(blocksPerGrid);
	dim3 bdim(threadsPerBlock);

	bool has_change = false;

	int *d_mat, *d_dist;
	bool *d_has_change, *d_has_negative_cycle;
	bool *relaxed_last_round, *relaxed_this_round;
	int *relaxed_times;

	hipMalloc(&d_mat, n * n * sizeof(int));
	hipMalloc(&d_dist, n * sizeof(int));
	hipMalloc(&d_has_change, sizeof(bool));
	hipMalloc(&d_has_negative_cycle, sizeof(bool));
	hipMalloc(&relaxed_last_round, n * sizeof(bool));
	hipMalloc(&relaxed_this_round, n * sizeof(bool));
	hipMalloc(&relaxed_times, n * sizeof(int));

	hipMemcpy(d_mat, mat, n * n * sizeof(int), hipMemcpyHostToDevice);
		
	relax_initial <<<gdim, bdim>>>(d_dist, d_has_negative_cycle, relaxed_last_round, relaxed_this_round, relaxed_times, n);

	while (true) {
		bf <<<gdim, bdim>>> (n, d_mat, d_dist, d_has_change, d_has_negative_cycle, relaxed_last_round, relaxed_this_round, relaxed_times);
		hipMemcpy(&has_change, d_has_change, sizeof(bool), hipMemcpyDeviceToHost);
		hipMemcpy(has_negative_cycle, d_has_negative_cycle, sizeof(bool), hipMemcpyDeviceToHost);
		if (!has_change || *has_negative_cycle)
			break;
		relax_swap <<<gdim, bdim>>>(relaxed_last_round, relaxed_this_round, n);
	}

	if (!*has_negative_cycle)
		hipMemcpy(dist, d_dist, sizeof(int) * n, hipMemcpyDeviceToHost);

	hipFree(d_mat);
	hipFree(d_dist);
	hipFree(d_has_change);
	hipFree(d_has_negative_cycle);
	hipFree(relaxed_last_round);
	hipFree(relaxed_this_round);
	hipFree(relaxed_times);
	//------end of your code------
}

int main(int argc, char **argv) {
	if (argc <= 1) {
		utils::abort_with_error_message("INPUT FILE WAS NOT FOUND!");
	}
	if (argc <= 3) {
		utils::abort_with_error_message("blocksPerGrid or threadsPerBlock WAS NOT FOUND!");
	}

	string filename = argv[1];
	int blockPerGrid = atoi(argv[2]);
	int threadsPerBlock = atoi(argv[3]);

	int *dist;
	bool has_negative_cycle = false;


	assert(utils::read_file(filename) == 0);
	dist = (int *)calloc(sizeof(int), utils::N);


	clock_t tb, te;
	tb = clock();
	//bellman-ford algorithm
	bellman_ford(blockPerGrid, threadsPerBlock, utils::N, utils::mat, dist, &has_negative_cycle);
	CHECK(hipDeviceSynchronize());
	te = clock();

	std::cerr.setf(std::ios::fixed);
	std::cerr << std::setprecision(6) << "Time(s): " << ((double)(te - tb) / CLOCKS_PER_SEC) << endl;
	utils::print_result(has_negative_cycle, dist);
	free(dist);
	free(utils::mat);

	return 0;
}
